#include "hip/hip_runtime.h"
//kernel - adding two matrices MatA and MatB

__global__ void MatAdd(float MatA[N][N], float MatB[N][N]), float MatC[N][N])
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if(i<N && j < N)
    {
        MatC[i][j]=MatA[i][j] + MatB[i][j];
    }
}

int main()
{
    //Matrix Addition kernel
    dim3 threadsPerBlock(16,16);
    dim3 numBlocks(N+threadsPerBlock.x-1)/threadsPerBlock.x, (N+threadsPerBlock.y-1)/threadsPerBlock.y);
    MatAdd<<<numBlocks, threadsPerBlock>>>(MatA, MatB, MatC)
}